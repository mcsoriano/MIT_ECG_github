/*
 * Copyright 2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//Source code adapted from CUDA Samples

//Include header (helper functions and CUSOLVE)
#include "../cuSolverDn_invMtM.h"
int linearSolverLU_MtM(int n, double *Acopy, int lda, double *b){
    hipsolverHandle_t handle;
    checkCudaErrors(hipsolverDnCreate(&handle));
    int bufferSize = 0;
    int *info = NULL;
    double *buffer = NULL;
    double *A = NULL;
    int *ipiv = NULL; // pivoting sequence
    int h_info = 0;
    double start, stop;
    double time_solve;
    hipEvent_t cstart, cstop;


    // 1 - Get Buffer
    checkCudaErrors(hipsolverDnDgetrf_bufferSize(handle, n, n, (double*)Acopy, lda, &bufferSize));

    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(double)*bufferSize));
    checkCudaErrors(hipMalloc(&A, sizeof(double)*lda*n));
    checkCudaErrors(hipMalloc(&ipiv, sizeof(int)*n));

    // prepare a copy of A because getrf will overwrite A with L
    checkCudaErrors(hipMemcpy(A, Acopy, sizeof(double)*lda*n, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

    int nIter=1;
    start = second();
    start = second();
    checkCudaErrors(hipEventCreate(&cstart));
    checkCudaErrors(hipEventCreate(&cstop));
    checkCudaErrors(hipEventRecord(cstart, NULL));

    // 2 - Factorize    
    for (int ni=0; ni<nIter; ni++){
    checkCudaErrors(hipsolverDnDgetrf(handle, n, n, A, lda, buffer, ipiv, info));
    checkCudaErrors(hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));
    if ( 0 != h_info ){
        printf("Error: LU factorization failed\n");
    }

    // 3 - Solve
    checkCudaErrors(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, n, 1, A, lda, ipiv, b, n, info));
    checkCudaErrors(hipDeviceSynchronize());

    }
    stop = second();
    checkCudaErrors(hipEventRecord(cstop, NULL));
    time_solve = 1./nIter * (stop - start);
    printf ("timing: LU = %10.6f sec\n", time_solve);
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, cstart, cstop));
    printf ("timing(msec): LU = %10.6f msec\n", msecTotal*1./nIter);

    //Store LU
    //double *hh_A = NULL; 
    //hh_A = (double*)malloc(sizeof(double)*n*n);
    //checkCudaErrors(hipMemcpy(hh_A, A, sizeof(double)*n*n, hipMemcpyDeviceToHost));
    //
    //printf(" ----------------- \n"); 
    //std::string fname="./lu_A.txt";
    //Write(fname, hh_A, n, n);
    //
    ////Free
    //if (hh_A  ) { free(hh_A);}


    if (info  ) { checkCudaErrors(hipFree(info  )); }
    if (buffer) { checkCudaErrors(hipFree(buffer)); }
    if (A     ) { checkCudaErrors(hipFree(A)); }
    if (ipiv  ) { checkCudaErrors(hipFree(ipiv));}

    return 0;
}


